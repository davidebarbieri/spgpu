#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */
 

#include "cudadebug.h"
#include "cudalang.h"

extern "C"
{
#include "core.h"
#include "ell.h"
}


#include "debug.h"


#ifdef ENABLE_CACHE
// Texture cache management
texture < int4, hipTextureType1D, hipReadModeElementType > x_tex;

#define bind_tex_x(x) hipBindTexture(NULL, x_tex, x)
#define unbind_tex_x(x) hipUnbindTexture(x_tex)
#endif

#define THREAD_BLOCK 128
#define MAX_N_FOR_A_CALL (THREAD_BLOCK*65535)

__device__ void
spgpuZellspmv_ridx (int i, hipDoubleComplex yVal, int outRow,
	hipDoubleComplex *z, const hipDoubleComplex *y, hipDoubleComplex alpha, const hipDoubleComplex* cM, const int* rP, int cMPitch, int rPPitch, const int* rS, int rows, const hipDoubleComplex *x, hipDoubleComplex beta, int baseIndex)
{
	hipDoubleComplex zProd = make_hipDoubleComplex(0.0, 0.0);

	rS += i; rP += i; cM += i;

	int rowSize = rS[0];

	for (int j = 0; j < rowSize / 2; j++)
	{
		int pointers1, pointers2;
		hipDoubleComplex values1, values2;
		hipDoubleComplex fetches1, fetches2;
		
		pointers1 = rP[0] - baseIndex;
		rP += rPPitch;  
		pointers2 = rP[0] - baseIndex;
		rP += rPPitch;  

		values1 = cM[0];
		cM += cMPitch;
		values2 = cM[0];
		cM += cMPitch;

#ifdef ENABLE_CACHE
		int4 f1 = tex1Dfetch (x_tex, pointers1);
		int4 f2 = tex1Dfetch (x_tex, pointers2);
		fetches1.x = __hiloint2double (f1.y, f1.x);
		fetches1.y = __hiloint2double (f1.w, f1.z);
		
		fetches2.x = __hiloint2double (f2.y, f2.x);
		fetches2.y = __hiloint2double (f2.w, f2.z);
#else
		fetches1 = x[pointers1];
		fetches2 = x[pointers2];

#endif

		zProd = hipCfma (values1, fetches1, zProd);
		zProd = hipCfma (values2, fetches2, zProd);	

	}

	// odd row size
	if (rowSize % 2)
	{
		int pointer = rP[0] - baseIndex;
		hipDoubleComplex value = cM[0];

		hipDoubleComplex fetch;
#ifdef ENABLE_CACHE
		int4 f1 = tex1Dfetch (x_tex, pointer);
		fetch.x = __hiloint2double (f1.y, f1.x);
		fetch.y = __hiloint2double (f1.w, f1.z);
#else
		fetch = x[pointer];
#endif
		zProd = hipCfma (value, fetch, zProd);
    }

	// Since z and y are accessed with the same offset by the same thread,
	// and the write to z follows the y read, y and z can share the same base address (in-place computing).
	if (cuDoubleComplex_isNotZero(beta))
		z[outRow] = hipCadd(hipCmul (beta, yVal), hipCmul (alpha, zProd));
	else
		z[outRow] = hipCmul(alpha, zProd);
}	


__global__ void
spgpuZellspmv_krn_ridx (hipDoubleComplex *z, const hipDoubleComplex *y, hipDoubleComplex alpha, const hipDoubleComplex* cM, const int* rP, int cMPitch, int rPPitch, const int* rS, const int* rIdx, int rows, const hipDoubleComplex *x, hipDoubleComplex beta, int baseIndex)
{
	int i = threadIdx.x + blockIdx.x * (THREAD_BLOCK);
	if (i >= rows)
		return;

	int outRow = rIdx[i];
	hipDoubleComplex yVal;
	if (cuDoubleComplex_isNotZero(beta))
		yVal = y[outRow];

	spgpuZellspmv_ridx (i, yVal, outRow,
		z, y, alpha, cM, rP, cMPitch, rPPitch, rS, rows, x, beta, baseIndex);
}


__device__ void
spgpuZellspmv_ (hipDoubleComplex *z, const hipDoubleComplex *y, hipDoubleComplex alpha, const hipDoubleComplex* cM, const int* rP, int cMPitch, int rPPitch, const int* rS, int rows, const hipDoubleComplex *x, hipDoubleComplex beta, int baseIndex)
{
	int i = threadIdx.x + blockIdx.x * (THREAD_BLOCK);
	if (i >= rows)
		return;

	hipDoubleComplex yVal;

	if (cuDoubleComplex_isNotZero(beta))
		yVal = y[i];

	spgpuZellspmv_ridx (i, yVal, i,
		z, y, alpha, cM, rP, cMPitch, rPPitch, rS, rows, x, beta, baseIndex);
}

// Force to recompile and optimize with llvm
__global__ void
spgpuZellspmv_krn_b0 (hipDoubleComplex *z, const hipDoubleComplex *y, hipDoubleComplex alpha, const hipDoubleComplex* cM, const int* rP, int cMPitch, int rPPitch, const int* rS, int rows, const hipDoubleComplex *x, int baseIndex)
{
	spgpuZellspmv_ (z, y, alpha, cM, rP, cMPitch, rPPitch, rS, rows, x, make_hipDoubleComplex(0.0,0.0), baseIndex);
}

__global__ void
spgpuZellspmv_krn (hipDoubleComplex *z, const hipDoubleComplex *y, hipDoubleComplex alpha, const hipDoubleComplex* cM, const int* rP, int cMPitch, int rPPitch, const int* rS, int rows, const hipDoubleComplex *x, hipDoubleComplex beta, int baseIndex)
{
	spgpuZellspmv_ (z, y, alpha, cM, rP, cMPitch, rPPitch, rS, rows, x, beta, baseIndex);
}

void
_spgpuZellspmv (spgpuHandle_t handle, hipDoubleComplex* z, const hipDoubleComplex *y, hipDoubleComplex alpha, const hipDoubleComplex* cM, const int* rP, int cMPitch, int rPPitch, const int* rS,  
	const __device int* rIdx, int rows, const hipDoubleComplex *x, hipDoubleComplex beta, int baseIndex)
{
	dim3 block (THREAD_BLOCK);
	dim3 grid ((rows + THREAD_BLOCK - 1) / THREAD_BLOCK);

#ifdef ENABLE_CACHE
	bind_tex_x ((const int4 *) x);
#endif

	if (rIdx)
		spgpuZellspmv_krn_ridx <<< grid, block, 0, handle->currentStream >>> (z, y, alpha, cM, rP, cMPitch, rPPitch, rS, rIdx, rows, x, beta, baseIndex);
	else
	{
		if (cuDoubleComplex_isNotZero(beta))
			spgpuZellspmv_krn <<< grid, block, 0, handle->currentStream >>> (z, y, alpha, cM, rP, cMPitch, rPPitch, rS, rows, x, beta, baseIndex);
		else
			spgpuZellspmv_krn_b0 <<< grid, block, 0, handle->currentStream >>> (z, y, alpha, cM, rP, cMPitch, rPPitch, rS, rows, x, baseIndex);
	}

#ifdef ENABLE_CACHE
  	unbind_tex_x ((const int4 *) x);
#endif

}


void 
spgpuZellspmv (spgpuHandle_t handle, 
	hipDoubleComplex* z, 
	const hipDoubleComplex *y, 
	hipDoubleComplex alpha, 
	const hipDoubleComplex* cM, 
	const int* rP, 
	int cMPitch, 
	int rPPitch, 
	const int* rS, 
	const __device int* rIdx, 
	int rows, 
	const hipDoubleComplex *x, 
	hipDoubleComplex beta, 
	int baseIndex)
{
	while (rows > MAX_N_FOR_A_CALL) //managing large vectors
	{
		_spgpuZellspmv (handle, z, y, alpha, cM, rP, cMPitch, rPPitch, rS, rIdx, MAX_N_FOR_A_CALL, x, beta, baseIndex);

		y = y + MAX_N_FOR_A_CALL;
		cM = cM + MAX_N_FOR_A_CALL;
		rP = rP + MAX_N_FOR_A_CALL;
		rS = rS + MAX_N_FOR_A_CALL;
		
		rows -= MAX_N_FOR_A_CALL;
	}
	
	_spgpuZellspmv (handle, z, y, alpha, cM, rP, cMPitch, rPPitch, rS, rIdx, rows, x, beta, baseIndex);
	cudaCheckError("CUDA error on ell_zspmv");
}

