#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */


#include "stdio.h"
#include "hip/hip_complex.h"


extern "C"
{
#include "core.h"
#include "vector.h"
}

#define BLOCK_SIZE 512
#define MAX_N_FOR_A_CALL (BLOCK_SIZE*65535)

// Single Precision Indexed Scatter
__global__ void siscat_gpu_kern(hipFloatComplex* vector, int count, const int* indexes, const hipFloatComplex* values, int firstIndex, hipFloatComplex beta)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	
	if (id < count)
	{	
		int pos = indexes[id]-firstIndex;
		
		if (cuFloatComplex_isNotZero(beta))
			vector[pos] = hipCfmaf(beta, vector[pos], values[id]);
		else
			vector[pos] = values[id];
	}
}

// Single Precision Indexed Gather
__global__ void sigath_gpu_kern(const hipFloatComplex* vector, int count, const int* indexes, hipFloatComplex* values, int firstIndex)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	
	if (id < count)
	{
		values[id] = vector[indexes[id]-firstIndex];
	}
}




void spgpuCscat_(spgpuHandle_t handle,
	__device hipFloatComplex* y,
	int xNnz,
	const __device hipFloatComplex *xValues,
	const __device int *xIndices,
	int xBaseIndex,
	hipFloatComplex beta)
{
	int msize = (xNnz+BLOCK_SIZE-1)/BLOCK_SIZE;

	dim3 block(BLOCK_SIZE);
	dim3 grid(msize);

	siscat_gpu_kern<<<grid, block, 0, handle->currentStream>>>(y, xNnz, xIndices, xValues, xBaseIndex, beta);
}

void spgpuCgath_(spgpuHandle_t handle,
	__device hipFloatComplex *xValues,
	int xNnz,
	const __device int *xIndices,
	int xBaseIndex,
	const __device hipFloatComplex* y)
{
	int msize = (xNnz+BLOCK_SIZE-1)/BLOCK_SIZE;

	dim3 block(BLOCK_SIZE);
	dim3 grid(msize);

	sigath_gpu_kern<<<grid, block, 0, handle->currentStream>>>(y, xNnz, xIndices, xValues, xBaseIndex);
}


void spgpuCscat(spgpuHandle_t handle,
	__device hipFloatComplex* y,
	int xNnz,
	const __device hipFloatComplex *xValues,
	const __device int *xIndices,
	int xBaseIndex, hipFloatComplex beta)
{
	while (xNnz > MAX_N_FOR_A_CALL) //managing large vectors
	{
		spgpuCscat_(handle, y, MAX_N_FOR_A_CALL, xValues, xIndices, xBaseIndex, beta);
	
		xIndices += MAX_N_FOR_A_CALL;
		xValues += MAX_N_FOR_A_CALL;
		xNnz -= MAX_N_FOR_A_CALL;
	}
	
	spgpuCscat_(handle, y, xNnz, xValues, xIndices, xBaseIndex, beta);
}	
	
void spgpuCgath(spgpuHandle_t handle,
	__device hipFloatComplex *xValues,
	int xNnz,
	const __device int *xIndices,
	int xBaseIndex,
	const __device hipFloatComplex* y)	
{
	while (xNnz > MAX_N_FOR_A_CALL) //managing large vectors
	{
		spgpuCgath_(handle, xValues, MAX_N_FOR_A_CALL, xIndices, xBaseIndex, y);
	
		xIndices += MAX_N_FOR_A_CALL;
		xValues += MAX_N_FOR_A_CALL;
		xNnz -= MAX_N_FOR_A_CALL;
	}
	
	spgpuCgath_(handle, xValues, xNnz, xIndices, xBaseIndex, y);
}
