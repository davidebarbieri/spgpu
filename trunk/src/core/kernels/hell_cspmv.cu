#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"

extern "C"
{
#include "core.h"
#include "hell.h"
}


#include "debug.h"


#ifdef ENABLE_CACHE
// Texture cache management
texture < hipFloatComplex, 1, hipReadModeElementType > x_tex;

#define bind_tex_x(x) hipBindTexture(NULL, x_tex, x)
#define unbind_tex_x(x) hipUnbindTexture(x_tex)
#endif

#define THREAD_BLOCK 128
#define MAX_N_FOR_A_CALL (THREAD_BLOCK*65535)

extern __shared__ int dynShrMem[]; 

__device__ void
spgpuChellspmv_ridx (int i, hipFloatComplex yVal, int outRow,
	hipFloatComplex *z, const hipFloatComplex *y, hipFloatComplex alpha, const hipFloatComplex* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, const int* rIdx, int rows, const hipFloatComplex *x, hipFloatComplex beta, int baseIndex)
{
	hipFloatComplex zProd = make_hipFloatComplex(0.0f, 0.0f);

	rS += i; 
	
	int hackId = i / hackSize;
	int hackLaneId = i % hackSize;

	// "volatile" used to avoid __syncthreads()
	volatile int* warpHackOffset = dynShrMem;

	int hackOffset;

	unsigned int laneId = threadIdx.x % 32;
	unsigned int warpId = threadIdx.x / 32;

	if (laneId == 0)
		warpHackOffset[warpId] = hackOffsets[hackId];
	
	hackOffset = warpHackOffset[warpId] + hackLaneId;

	rP += hackOffset; 
	cM += hackOffset; 

	int rowSize = rS[0];

	for (int j = 0; j < rowSize / 2; j++)
	{
		int pointers1, pointers2;
		hipFloatComplex values1, values2, fetches1, fetches2;
		
		pointers1 = rP[0] - baseIndex;
		rP += hackSize;  
		pointers2 = rP[0] - baseIndex;
		rP += hackSize;

		values1 = cM[0];
		cM += hackSize;
		values2 = cM[0];
		cM += hackSize;

#ifdef ENABLE_CACHE
		fetches1 = tex1Dfetch (x_tex, pointers1);
		fetches2 = tex1Dfetch (x_tex, pointers2);
#else
		fetches1 = x[pointers1];
		fetches2 = x[pointers2];
#endif

		// avoid MAD on pre-Fermi
		zProd = hipCfmaf (values1, fetches1, zProd);
		zProd = hipCfmaf (values2, fetches2, zProd);	
	}

	// odd row size
	if (rowSize % 2)
    	{
		int pointer = rP[0] - baseIndex;
		hipFloatComplex value = cM[0];
		hipFloatComplex fetch;

#ifdef ENABLE_CACHE
		fetch = tex1Dfetch (x_tex, pointer);
#else
		fetch = x[pointer];
#endif
		zProd = hipCfmaf (value, fetch, zProd);
    }

	// Since z and y are accessed with the same offset by the same thread,
	// and the write to z follows the y read, y and z can share the same base address (in-place computing).
	if (cuFloatComplex_isNotZero(beta))
		z[outRow] = hipCfmaf(beta, yVal, hipCmulf(alpha, zProd));
	else
		z[outRow] = hipCmulf(alpha, zProd);
}

__global__ void
spgpuChellspmv_krn_ridx (hipFloatComplex *z, const hipFloatComplex *y, hipFloatComplex alpha, const hipFloatComplex* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, const int* rIdx, int rows, const hipFloatComplex *x, hipFloatComplex beta, int baseIndex)
{
	int i = threadIdx.x + blockIdx.x * (THREAD_BLOCK);
	if (i >= rows)
		return;

	int outRow = rIdx[i];
	hipFloatComplex yVal;
	if (cuFloatComplex_isNotZero(beta))
		yVal = y[outRow];

	spgpuChellspmv_ridx (i, yVal, outRow,
		z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rIdx, rows, x, beta, baseIndex);
}

__device__ void
spgpuChellspmv_ (hipFloatComplex *z, const hipFloatComplex *y, hipFloatComplex alpha, const hipFloatComplex* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, int rows, const hipFloatComplex *x, hipFloatComplex beta, int baseIndex)
{
	int i = threadIdx.x + blockIdx.x * (THREAD_BLOCK);
	if (i >= rows)
		return;

	hipFloatComplex yVal;

	if (cuFloatComplex_isNotZero(beta))
		yVal = y[i];

	spgpuChellspmv_ridx (i, yVal, i,
		z, y, alpha, cM, rP, hackSize, hackOffsets, rS, NULL, rows, x, beta, baseIndex);
}

// Force to recompile and optimize with llvm
__global__ void
spgpuChellspmv_krn_b0 (hipFloatComplex *z, const hipFloatComplex *y, hipFloatComplex alpha, const hipFloatComplex* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, int rows, const hipFloatComplex *x, int baseIndex)
{
	spgpuChellspmv_ (z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rows, x, make_hipFloatComplex(0.0f, 0.0f), baseIndex);
}

__global__ void
spgpuChellspmv_krn (hipFloatComplex *z, const hipFloatComplex *y, hipFloatComplex alpha, const hipFloatComplex* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, int rows, const hipFloatComplex *x, hipFloatComplex beta, int baseIndex)
{
	spgpuChellspmv_ (z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rows, x, beta, baseIndex);
}

void
_spgpuChellspmv (spgpuHandle_t handle, hipFloatComplex* z, const hipFloatComplex *y, hipFloatComplex alpha, const hipFloatComplex* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, const int* rIdx, int rows, const hipFloatComplex *x, hipFloatComplex beta, int baseIndex)
{
	dim3 block (THREAD_BLOCK);
	dim3 grid ((rows + THREAD_BLOCK - 1) / THREAD_BLOCK);

	int warpsPerBlock = THREAD_BLOCK/handle->warpSize;

#ifdef ENABLE_CACHE
	bind_tex_x (x);
#endif

	
	if (rIdx)
		spgpuChellspmv_krn_ridx <<< grid, block, warpsPerBlock*sizeof(int), handle->currentStream >>> (z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rIdx, rows, x, beta, baseIndex);
	else
	{
		if (cuFloatComplex_isNotZero(beta))
			spgpuChellspmv_krn <<< grid, block, warpsPerBlock*sizeof(int), handle->currentStream >>> (z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rows, x, beta, baseIndex);
		else
			spgpuChellspmv_krn_b0 <<< grid, block, warpsPerBlock*sizeof(int), handle->currentStream >>> (z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rows, x, baseIndex);
	}
	
#ifdef ENABLE_CACHE
  	unbind_tex_x (x);
#endif

}



void 
spgpuChellspmv (spgpuHandle_t handle,
	__device hipFloatComplex *z, 
	const __device hipFloatComplex *y, 
	hipFloatComplex alpha, 
	const __device hipFloatComplex* cM, 
	const __device int* rP,
	int hackSize,
	const __device int* hackOffsets, 
	const __device int* rS,
	const __device int* rIdx, 
	int rows, 
	const __device hipFloatComplex *x, 
	hipFloatComplex beta,
	int baseIndex)
{
	__assert(hackSize % 32 == 0, "Only hacks whose length is a multiple of 32 are supported...");

	
	while (rows > MAX_N_FOR_A_CALL) //managing large vectors
	{
		_spgpuChellspmv (handle, z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rIdx, MAX_N_FOR_A_CALL, x, beta, baseIndex);

		y = y + MAX_N_FOR_A_CALL;
		z = z + MAX_N_FOR_A_CALL;
		rS = rS + MAX_N_FOR_A_CALL;
		hackOffsets += MAX_N_FOR_A_CALL/hackSize;
		
		rows -= MAX_N_FOR_A_CALL;
	}
	
	_spgpuChellspmv (handle, z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rIdx, rows, x, beta, baseIndex);
	
	cudaCheckError("CUDA error on hell_cspmv");
}
