#include "hip/hip_runtime.h"
#include "vector.h"
#include "stdio.h"

#define BLOCK_SIZE 512
#define MAX_N_FOR_A_CALL (BLOCK_SIZE*65535)

// Single Precision Indexed Scatter
__global__ void siscat_gpu_kern(float* vector, int count, const int* indexes, const float* values, int firstIndex)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	
	if (id < count)
	{	
		vector[indexes[id]-firstIndex] = values[id];
	}
}

// Single Precision Indexed Gather
__global__ void sigath_gpu_kern(const float* vector, int count, const int* indexes, float* values, int firstIndex)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	
	if (id < count)
	{
		values[id] = vector[indexes[id]-firstIndex];
	}
}




void spgpuSscat_(spgpuHandle_t handle,
	__device float* y,
	int xNnz,
	const __device float *xValues,
	const __device int *xIndices,
	int xBaseIndex)
{
	int msize = (xNnz+BLOCK_SIZE-1)/BLOCK_SIZE;

	dim3 block(BLOCK_SIZE);
	dim3 grid(msize);

	siscat_gpu_kern<<<grid, block, 0, handle->currentStream>>>(y, xNnz, xIndices, xValues, xBaseIndex);
}

void spgpuSgath_(spgpuHandle_t handle,
	__device float *xValues,
	int xNnz,
	const __device int *xIndices,
	int xBaseIndex,
	const __device float* y)
{
	int msize = (xNnz+BLOCK_SIZE-1)/BLOCK_SIZE;

	dim3 block(BLOCK_SIZE);
	dim3 grid(msize);

	sigath_gpu_kern<<<grid, block, 0, handle->currentStream>>>(y, xNnz, xIndices, xValues, xBaseIndex);
}


void spgpuSscat(spgpuHandle_t handle,
	__device float* y,
	int xNnz,
	const __device float *xValues,
	const __device int *xIndices,
	int xBaseIndex)
{
	while (xNnz > MAX_N_FOR_A_CALL) //managing large vectors
	{
		spgpuSscat_(handle, y, MAX_N_FOR_A_CALL, xValues, xIndices, xBaseIndex);
	
		xIndices += MAX_N_FOR_A_CALL;
		xValues += MAX_N_FOR_A_CALL;
		xNnz -= MAX_N_FOR_A_CALL;
	}
	
	spgpuSscat_(handle, y, xNnz, xValues, xIndices, xBaseIndex);
}	
	
void spgpuSgath(spgpuHandle_t handle,
	__device float *xValues,
	int xNnz,
	const __device int *xIndices,
	int xBaseIndex,
	const __device float* y)	
{
	while (xNnz > MAX_N_FOR_A_CALL) //managing large vectors
	{
		spgpuSgath_(handle, xValues, MAX_N_FOR_A_CALL, xIndices, xBaseIndex, y);
	
		xIndices += MAX_N_FOR_A_CALL;
		xValues += MAX_N_FOR_A_CALL;
		xNnz -= MAX_N_FOR_A_CALL;
	}
	
	spgpuSgath_(handle, xValues, xNnz, xIndices, xBaseIndex, y);
}
