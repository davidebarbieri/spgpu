#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */


#include "stdio.h"

extern "C"
{
#include "core.h"
#include "vector.h"
}

#define BLOCK_SIZE 512
#define MAX_N_FOR_A_CALL (BLOCK_SIZE*65535)

// Single Precision Indexed Scatter
__global__ void siscat_gpu_kern(float* vector, int count, const int* indexes, const float* values, int firstIndex, float beta)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	
	if (id < count)
	{	
		int pos = indexes[id]-firstIndex;
		vector[pos] = beta*vector[pos]+values[id];
	}
}

// Single Precision Indexed Gather
__global__ void sigath_gpu_kern(const float* vector, int count, const int* indexes, float* values, int firstIndex)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	
	if (id < count)
	{
		values[id] = vector[indexes[id]-firstIndex];
	}
}




void spgpuSscat_(spgpuHandle_t handle,
	__device float* y,
	int xNnz,
	const __device float *xValues,
	const __device int *xIndices,
	int xBaseIndex,
	float beta)
{
	int msize = (xNnz+BLOCK_SIZE-1)/BLOCK_SIZE;

	dim3 block(BLOCK_SIZE);
	dim3 grid(msize);

	siscat_gpu_kern<<<grid, block, 0, handle->currentStream>>>(y, xNnz, xIndices, xValues, xBaseIndex, beta);
}

void spgpuSgath_(spgpuHandle_t handle,
	__device float *xValues,
	int xNnz,
	const __device int *xIndices,
	int xBaseIndex,
	const __device float* y)
{
	int msize = (xNnz+BLOCK_SIZE-1)/BLOCK_SIZE;

	dim3 block(BLOCK_SIZE);
	dim3 grid(msize);

	sigath_gpu_kern<<<grid, block, 0, handle->currentStream>>>(y, xNnz, xIndices, xValues, xBaseIndex);
}


void spgpuSscat(spgpuHandle_t handle,
	__device float* y,
	int xNnz,
	const __device float *xValues,
	const __device int *xIndices,
	int xBaseIndex, float beta)
{
	while (xNnz > MAX_N_FOR_A_CALL) //managing large vectors
	{
		spgpuSscat_(handle, y, MAX_N_FOR_A_CALL, xValues, xIndices, xBaseIndex, beta);
	
		xIndices += MAX_N_FOR_A_CALL;
		xValues += MAX_N_FOR_A_CALL;
		xNnz -= MAX_N_FOR_A_CALL;
	}
	
	spgpuSscat_(handle, y, xNnz, xValues, xIndices, xBaseIndex, beta);
}	
	
void spgpuSgath(spgpuHandle_t handle,
	__device float *xValues,
	int xNnz,
	const __device int *xIndices,
	int xBaseIndex,
	const __device float* y)	
{
	while (xNnz > MAX_N_FOR_A_CALL) //managing large vectors
	{
		spgpuSgath_(handle, xValues, MAX_N_FOR_A_CALL, xIndices, xBaseIndex, y);
	
		xIndices += MAX_N_FOR_A_CALL;
		xValues += MAX_N_FOR_A_CALL;
		xNnz -= MAX_N_FOR_A_CALL;
	}
	
	spgpuSgath_(handle, xValues, xNnz, xIndices, xBaseIndex, y);
}
