#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"

extern "C"
{
#include "core.h"
#include "hell.h"
}


#include "debug.h"


#ifdef ENABLE_CACHE
// Texture cache management
texture < float, 1, hipReadModeElementType > x_tex;

#define bind_tex_x(x) hipBindTexture(NULL, x_tex, x)
#define unbind_tex_x(x) hipUnbindTexture(x_tex)
#endif

#define THREAD_BLOCK 128
#define MAX_N_FOR_A_CALL (THREAD_BLOCK*65535)

extern __shared__ int dynShrMem[]; 

__global__ void
spgpuShellspmv_krn (float *z, const float *y, float alpha, const float* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, const int* rIdx, int rows, const float *x, float beta, int baseIndex)
{
	int i = threadIdx.x + blockIdx.x * (THREAD_BLOCK);

	if (i >= rows)
		return;

	float yVal;
	
	int outRow;

	if (rIdx)
		outRow = rIdx[i];
	else
		outRow = i;

	if (beta != 0.0f)
	{
		yVal = y[outRow];
	}

	float zProd = 0.0f;

	rS += i; 
	
	int hackId = i / hackSize;
	int hackLaneId = i % hackSize;

	// "volatile" used to avoid __syncthreads()
	volatile int* warpHackOffset = dynShrMem;

	int hackOffset;

	unsigned int laneId = threadIdx.x % 32;
	unsigned int warpId = threadIdx.x / 32;

	if (laneId == 0)
		warpHackOffset[warpId] = hackOffsets[hackId];
	
	hackOffset = warpHackOffset[warpId] + hackLaneId;

	rP += hackOffset; 
	cM += hackOffset; 

	int rowSize = rS[0];

	for (int j = 0; j < rowSize / 2; j++)
	{
		int pointers1, pointers2;
		float values1, values2, fetches1, fetches2;
		
		pointers1 = rP[0] - baseIndex;
		rP += hackSize;  
		pointers2 = rP[0] - baseIndex;
		rP += hackSize;

		values1 = cM[0];
		cM += hackSize;
		values2 = cM[0];
		cM += hackSize;

#ifdef ENABLE_CACHE
		fetches1 = tex1Dfetch (x_tex, pointers1);
		fetches2 = tex1Dfetch (x_tex, pointers2);
#else
		fetches1 = x[pointers1];
		fetches2 = x[pointers2];
#endif

		// avoid MAD on pre-Fermi
		zProd = PREC_FADD(zProd, PREC_FMUL (values1, fetches1));
		zProd = PREC_FADD(zProd, PREC_FMUL (values2, fetches2));	
	}

	// odd row size
	if (rowSize % 2)
    {
      int pointer = rP[0] - baseIndex;
      float value = cM[0];
      float fetch;

#ifdef ENABLE_CACHE
      fetch = tex1Dfetch (x_tex, pointer);
#else
      fetch = x[pointer];
#endif

      zProd = PREC_FADD(zProd, PREC_FMUL (value, fetch));
    }

	if (beta == 0.0f)
		z[outRow] = PREC_FMUL(alpha, zProd);
	else
		z[outRow] = PREC_FADD(PREC_FMUL (beta, yVal), PREC_FMUL (alpha, zProd));
}


void
_spgpuShellspmv (spgpuHandle_t handle, float* z, const float *y, float alpha, const float* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, const int* rIdx, int rows, const float *x, float beta, int baseIndex)
{
	dim3 block (THREAD_BLOCK);
	dim3 grid ((rows + THREAD_BLOCK - 1) / THREAD_BLOCK);

	int warpsPerBlock = THREAD_BLOCK/handle->warpSize;

#ifdef ENABLE_CACHE
	bind_tex_x (x);
#endif

	spgpuShellspmv_krn <<< grid, block, warpsPerBlock*sizeof(int), handle->currentStream >>> (z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rIdx, rows, x, beta, baseIndex);
	
#ifdef ENABLE_CACHE
  	unbind_tex_x (x);
#endif

	cudaCheckError("ERRORE (SSPVM)!");
}



void 
spgpuShellspmv (spgpuHandle_t handle,
	__device float *z, 
	const __device float *y, 
	float alpha, 
	const __device float* cM, 
	const __device int* rP,
	int hackSize,
	const __device int* hackOffsets, 
	const __device int* rS,
	const __device int* rIdx, 
	int rows, 
	const __device float *x, 
	float beta,
	int baseIndex)
{
	__assert(hackSize % 32 == 0, "Only hacks whose length is a multiple of 32 are supported...");

	
	while (rows > MAX_N_FOR_A_CALL) //managing large vectors
	{
		_spgpuShellspmv (handle, z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rIdx, MAX_N_FOR_A_CALL, x, beta, baseIndex);

		y = y + MAX_N_FOR_A_CALL;
		rS = rS + MAX_N_FOR_A_CALL;
		hackOffsets += MAX_N_FOR_A_CALL/hackSize;
		
		rows -= MAX_N_FOR_A_CALL;
	}
	
	_spgpuShellspmv (handle, z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rIdx, rows, x, beta, baseIndex);
	
	cudaCheckError("CUDA error on sspmv");
}
