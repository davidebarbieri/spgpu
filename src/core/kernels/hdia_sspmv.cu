#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2013
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */
 
#include "cudadebug.h"
#include "cudalang.h"

extern "C"
{
#include "core.h"
#include "hdia.h"
}

#include "debug.h"

#ifdef ENABLE_CACHE
// Texture cache management
texture < float, 1, hipReadModeElementType > x_tex;

#define bind_tex_x(x) hipBindTexture(NULL, x_tex, x)
#define unbind_tex_x(x) hipUnbindTexture(x_tex)
#endif

__device__ void
spgpuShdiaspmv_ (float *z, const float *y, float alpha, const float* dM, const int* offsets, int hackSize, const int* hackOffsets, 
	int rows, int cols, const float *x, float beta, int hackCount)
{
	int i = threadIdx.x + blockIdx.x * (blockDim.x);
	
	float yVal = 0.0f;

	if (i < rows && beta != 0.0f)
		yVal = y[i];

	float zProd = 0.0f;
	
	int hackId = i / hackSize;
	int hackLaneId = i % hackSize;
	
	
	// shared between offsetsChunks and warpHackOffsetTemp
	extern __shared__ int dynShrMem[]; 

	int hackOffset = 0;
	int nextOffset = 0;
	
	unsigned int laneId = threadIdx.x % warpSize;
	unsigned int warpId = threadIdx.x / warpSize;
	
#if __CUDA_ARCH__ < 300	
	int* warpHackOffset = dynShrMem;


	if (laneId == 0 && i < rows)
	{
		warpHackOffset[warpId] = hackOffsets[hackId];
		warpHackOffset[warpId + (blockDim.x / warpSize)] = hackOffsets[hackId+1];
	}
	
	__syncthreads();
	hackOffset = warpHackOffset[warpId];
	nextOffset = warpHackOffset[warpId + blockDim.x / warpSize];
	__syncthreads();
#else
	if (laneId == 0 && i < rows)
	{
		hackOffset = hackOffsets[hackId];
		nextOffset = hackOffsets[hackId+1];
	}
	
	hackOffset = __shfl(hackOffset, 0);	
	nextOffset = __shfl(nextOffset, 0);
#endif
	
	if (hackId >= hackCount)
		return;

	dM += hackOffset*hackSize + hackLaneId;
	offsets += hackOffset;
	
	// diags for this hack is next hackOffset minus current hackOffset
	int diags = nextOffset - hackOffset;
	
	
	// Warp oriented
	int rounds = (diags + warpSize - 1)/warpSize;
	
	volatile int *offsetsChunk = dynShrMem + warpId*warpSize;
	
	for (int r = 0; r < rounds; r++)
	{
		// in the last round diags will be <= warpSize
		if (laneId < diags)
			offsetsChunk[laneId] = offsets[laneId];
	
		if (i < rows)
		{
			int count = min(diags, warpSize);
			
			int j;
			for (j=0; j<=count-2; j += 2)
			{
				// prefetch 3 values
				int column1 = offsetsChunk[j] + i;
				int column2 = offsetsChunk[j+1] + i;			
				
				float xValue1, xValue2;
				float mValue1, mValue2;
				
				bool inside1 = column1 >= 0 && column1 < cols;
				bool inside2 = column2 >= 0 && column2 < cols;
				
				if(inside1)
                		{
                			mValue1 = dM[0];
#ifdef ENABLE_CACHE
					xValue1 = tex1Dfetch (x_tex, column1);
#else
					xValue1 = x[column1];
#endif				
				}
				
				dM += hackSize;
							
				if(inside2)
                		{
                			mValue2 = dM[0];
#ifdef ENABLE_CACHE
					xValue2 = tex1Dfetch (x_tex, column2);
#else
					xValue2 = x[column2];
#endif					
				}

				dM += hackSize;					
											
				if(inside1)
					zProd = PREC_FADD(zProd, PREC_FMUL (xValue1, mValue1));
				
				if(inside2)
					zProd = PREC_FADD(zProd, PREC_FMUL (xValue2, mValue2));
			}
	
			for (;j<count; ++j)
			{
				int column = offsetsChunk[j] + i;
				
				if(column >= 0 && column < cols)
                		{
#ifdef ENABLE_CACHE
					float xValue = tex1Dfetch (x_tex, column);
#else
					float xValue = x[column];
#endif				
			
					zProd = PREC_FADD(zProd, PREC_FMUL (xValue, dM[0]));
				}
				
				dM += hackSize;
			}
		}
		
		diags -= warpSize;
		offsets += warpSize;
	}


	// Since z and y are accessed with the same offset by the same thread,
	// and the write to z follows the y read, y and z can share the same base address (in-place computing).
	
	if (i >= rows)
		return;
	
	if (beta == 0.0f)
		z[i] = PREC_FMUL(alpha, zProd);
	else
		z[i] = PREC_FADD(PREC_FMUL (beta, yVal), PREC_FMUL (alpha, zProd));
}

// Force to recompile and optimize with llvm
__global__ void
spgpuShdiaspmv_krn_b0 (float *z, const float *y, float alpha, const float* dM, const int* offsets, int hackSize, const int* hackOffsets, int rows, int cols, const float *x, int hackCount)
{
	spgpuShdiaspmv_ (z, y, alpha, dM, offsets, hackSize, hackOffsets, rows, cols, x, 0.0f, hackCount);
}

__global__ void
spgpuShdiaspmv_krn (float *z, const float *y, float alpha, const float* dM, const int* offsets, int hackSize, const int* hackOffsets, int rows, int cols, const float *x, float beta, int hackCount)
{
	spgpuShdiaspmv_ (z, y, alpha, dM, offsets, hackSize, hackOffsets, rows, cols, x, beta, hackCount);
}

void
_spgpuShdiaspmv (spgpuHandle_t handle, int threadCount, float* z, const float *y, float alpha, 
	const float* dM, const int* offsets, int hackSize, const int* hackOffsets, int rows, int cols,
	const float *x, float beta)
{
	dim3 block (threadCount);
	dim3 grid ((rows + threadCount - 1) / threadCount);

	int hackCount = (rows + hackSize - 1)/hackSize;
	
#ifdef ENABLE_CACHE
	bind_tex_x (x);
#endif

	if (beta != 0.0f)
		spgpuShdiaspmv_krn <<< grid, block, block.x*sizeof(int), handle->currentStream >>> (z, y, alpha, dM, offsets, hackSize, hackOffsets, rows, cols, x, beta, hackCount);
	else
		spgpuShdiaspmv_krn_b0 <<< grid, block, block.x*sizeof(int), handle->currentStream >>> (z, y, alpha, dM, offsets, hackSize, hackOffsets, rows, cols, x, hackCount);

#ifdef ENABLE_CACHE
  	unbind_tex_x (x);
#endif

}

void 
spgpuShdiaspmv (spgpuHandle_t handle, 
	float* z, 
	const float *y, 
	float alpha, 
	const float* dM, 
	const int* offsets, 
	int hackSize, 
	const int* hackOffsets,
	int rows,
	int cols, 
	const float *x, 
	float beta)
{
	__assert(hackSize % 32 == 0, "Only hacks whose length is a multiple of 32 are supported...");
	
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(spgpuShdiaspmv_krn), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(spgpuShdiaspmv_krn_b0), hipFuncCachePreferL1);
	
	hipDeviceProp_t deviceProp;
    	hipGetDeviceProperties(&deviceProp, 0);
    	
    	int threadCount = 128; 

	int maxNForACall = max(handle->maxGridSizeX, threadCount*handle->maxGridSizeX);

	// maxNForACall should be a multiple of hackSize
	maxNForACall = (maxNForACall/hackSize)*hackSize;
	
	while (rows > maxNForACall) //managing large vectors
	{
		_spgpuShdiaspmv (handle, threadCount, z, y, alpha, dM, offsets, hackSize, hackOffsets, maxNForACall, cols, x, beta);

		y = y + maxNForACall;
		z = z + maxNForACall;
		hackOffsets += maxNForACall/hackSize;
		
		rows -= maxNForACall;
	}
	
	_spgpuShdiaspmv (handle, threadCount, z, y, alpha, dM, offsets, hackSize, hackOffsets, rows, cols, x, beta);
	
	cudaCheckError("CUDA error on hdia_sspmv");
}

