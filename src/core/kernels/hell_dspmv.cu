#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"

extern "C"
{
#include "core.h"
#include "hell.h"
}


#include "debug.h"


#ifdef ENABLE_CACHE
// Texture cache management
texture < int2, 1, hipReadModeElementType > x_tex;

#define bind_tex_x(x) hipBindTexture(NULL, x_tex, x)
#define unbind_tex_x(x) hipUnbindTexture(x_tex)
#endif

#define THREAD_BLOCK 128
#define MAX_N_FOR_A_CALL (THREAD_BLOCK*65535)

extern __shared__ int dynShrMem[]; 

__global__ void
spgpuDhellspmv_krn (double *z, const double *y, double alpha, const double* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, const int* rIdx, int rows, const double *x, double beta, int baseIndex)
{
	int i = threadIdx.x + blockIdx.x * (THREAD_BLOCK);

	if (i >= rows)
		return;

	double yVal;
	
	int outRow;

	if (rIdx)
		outRow = rIdx[i];
	else
		outRow = i;

	if (beta != 0.0)
	{
		yVal = y[outRow];
	}

	double zProd = 0.0;

	rS += i; 
	
	int hackId = i / hackSize;
	int hackLaneId = i % hackSize;

	// "volatile" used to avoid __syncthreads()
	volatile int* warpHackOffset = dynShrMem;

	int hackOffset;

	unsigned int laneId = threadIdx.x % 32;
	unsigned int warpId = threadIdx.x / 32;

	if (laneId == 0)
		warpHackOffset[warpId] = hackOffsets[hackId];
	
	hackOffset = warpHackOffset[warpId] + hackLaneId;

	rP += hackOffset; 
	cM += hackOffset; 

	int rowSize = rS[0];

	for (int j = 0; j < rowSize / 2; j++)
	{
		int pointers1, pointers2;
		double values1, values2;
		
#ifdef ENABLE_CACHE
		int2 fetches1, fetches2;
#else
		double fetches1, fetches2;
#endif
		
		pointers1 = rP[0] - baseIndex;
		rP += hackSize;  
		pointers2 = rP[0] - baseIndex;
		rP += hackSize;

		values1 = cM[0];
		cM += hackSize;
		values2 = cM[0];
		cM += hackSize;

#ifdef ENABLE_CACHE

		fetches1 = tex1Dfetch (x_tex, pointers1);
		fetches2 = tex1Dfetch (x_tex, pointers2);

		// avoid MAD on pre-Fermi
		zProd = PREC_DADD(zProd, PREC_DMUL (values1, __hiloint2double (fetches1.y, fetches1.x)));
		zProd = PREC_DADD(zProd, PREC_DMUL (values2, __hiloint2double (fetches2.y, fetches2.x)));
#else
		fetches1 = x[pointers1];
		fetches2 = x[pointers2];

		// avoid MAD on pre-Fermi
		zProd = PREC_DADD(zProd, PREC_DMUL (values1, fetches1));
		zProd = PREC_DADD(zProd, PREC_DMUL (values2, fetches2));	
#endif
	}

	// odd row size
	if (rowSize % 2)
    {
      int pointer = rP[0] - baseIndex;
      double value = cM[0];

#ifdef ENABLE_CACHE
		int2 fetch;
		fetch = tex1Dfetch (x_tex, pointer);
		zProd = PREC_DADD(zProd, PREC_DMUL (value, __hiloint2double (fetch.y, fetch.x)));
#else
		double fetch;
		fetch = x[pointer];
		zProd = PREC_DADD(zProd, PREC_DMUL (value, fetch));
#endif
    }

	if (beta == 0.0)
		z[outRow] = PREC_DMUL(alpha, zProd);
	else
		z[outRow] = PREC_DADD(PREC_DMUL (beta, yVal), PREC_DMUL (alpha, zProd));
}


void
_spgpuDhellspmv (spgpuHandle_t handle, double* z, const double *y, double alpha, const double* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, const int* rIdx, int rows, const double *x, double beta, int baseIndex)
{
	dim3 block (THREAD_BLOCK);
	dim3 grid ((rows + THREAD_BLOCK - 1) / THREAD_BLOCK);

	int warpsPerBlock = THREAD_BLOCK/handle->warpSize;

#ifdef ENABLE_CACHE
	bind_tex_x ((const int2 *) x);
#endif

	spgpuDhellspmv_krn <<< grid, block, warpsPerBlock*sizeof(int), handle->currentStream >>> (z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rIdx, rows, x, beta, baseIndex);
	
#ifdef ENABLE_CACHE
  	unbind_tex_x ((const int2 *) x);
#endif

	cudaCheckError("ERRORE (SSPVM)!");
}



void 
spgpuDhellspmv (spgpuHandle_t handle,
	__device double *z, 
	const __device double *y, 
	double alpha, 
	const __device double* cM, 
	const __device int* rP,
	int hackSize,
	const __device int* hackOffsets, 
	const __device int* rS,
	const __device int* rIdx, 
	int rows, 
	const __device double *x, 
	double beta,
	int baseIndex)
{
	__assert(hackSize % 32 == 0, "Only hacks whose length is a multiple of 32 are supported...");

	
	while (rows > MAX_N_FOR_A_CALL) //managing large vectors
	{
		_spgpuDhellspmv (handle, z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rIdx, MAX_N_FOR_A_CALL, x, beta, baseIndex);

		y = y + MAX_N_FOR_A_CALL;
		rS = rS + MAX_N_FOR_A_CALL;
		hackOffsets += MAX_N_FOR_A_CALL/hackSize;
		
		rows -= MAX_N_FOR_A_CALL;
	}
	
	_spgpuDhellspmv (handle, z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rIdx, rows, x, beta, baseIndex);
	
	cudaCheckError("CUDA error on sspmv");
}
