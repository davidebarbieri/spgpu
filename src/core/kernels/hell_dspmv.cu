#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"

extern "C"
{
#include "core.h"
#include "hell.h"
}


#include "debug.h"


#ifdef ENABLE_CACHE
// Texture cache management
texture < int2, 1, hipReadModeElementType > x_tex;

#define bind_tex_x(x) hipBindTexture(NULL, x_tex, x)
#define unbind_tex_x(x) hipUnbindTexture(x_tex)
#endif

#define THREAD_BLOCK 128
#define MAX_N_FOR_A_CALL (THREAD_BLOCK*65535)

#if __CUDA_ARCH__ < 300
extern __shared__ int dynShrMem[]; 
#endif

__device__ void
spgpuDhellspmv_ridx (int i, double yVal, int outRow,
	double *z, const double *y, double alpha, const double* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, const int* rIdx, int rows, const double *x, double beta, int baseIndex)
{
	

	double zProd = 0.0;

	rS += i; 
	
	int hackId = i / hackSize;
	int hackLaneId = i % hackSize;

	int hackOffset;
	unsigned int laneId = threadIdx.x % 32;
#if __CUDA_ARCH__ < 300
        // "volatile" used to avoid __syncthreads()
        volatile int* warpHackOffset = dynShrMem;

        unsigned int warpId = threadIdx.x / 32;

        if (laneId == 0)
                warpHackOffset[warpId] = hackOffsets[hackId];

        hackOffset = warpHackOffset[warpId] + hackLaneId;
#else
     	if (laneId == 0)
                hackOffset = hackOffsets[hackId];
        hackOffset = __shfl(hackOffset, 0) + hackLaneId;

#endif

	rP += hackOffset; 
	cM += hackOffset; 

	int rowSize = rS[0];

	for (int j = 0; j < rowSize / 2; j++)
	{
		int pointers1, pointers2;
		double values1, values2;
		
#ifdef ENABLE_CACHE
		int2 fetches1, fetches2;
#else
		double fetches1, fetches2;
#endif
		
		pointers1 = rP[0] - baseIndex;
		rP += hackSize;  
		pointers2 = rP[0] - baseIndex;
		rP += hackSize;

		values1 = cM[0];
		cM += hackSize;
		values2 = cM[0];
		cM += hackSize;

#ifdef ENABLE_CACHE

		fetches1 = tex1Dfetch (x_tex, pointers1);
		fetches2 = tex1Dfetch (x_tex, pointers2);

		// avoid MAD on pre-Fermi
		zProd = PREC_DADD(zProd, PREC_DMUL (values1, __hiloint2double (fetches1.y, fetches1.x)));
		zProd = PREC_DADD(zProd, PREC_DMUL (values2, __hiloint2double (fetches2.y, fetches2.x)));
#else
		fetches1 = x[pointers1];
		fetches2 = x[pointers2];

		// avoid MAD on pre-Fermi
		zProd = PREC_DADD(zProd, PREC_DMUL (values1, fetches1));
		zProd = PREC_DADD(zProd, PREC_DMUL (values2, fetches2));	
#endif
	}

	// odd row size
	if (rowSize % 2)
    {
      int pointer = rP[0] - baseIndex;
      double value = cM[0];

#ifdef ENABLE_CACHE
		int2 fetch;
		fetch = tex1Dfetch (x_tex, pointer);
		zProd = PREC_DADD(zProd, PREC_DMUL (value, __hiloint2double (fetch.y, fetch.x)));
#else
		double fetch;
		fetch = x[pointer];
		zProd = PREC_DADD(zProd, PREC_DMUL (value, fetch));
#endif
    }

	// Since z and y are accessed with the same offset by the same thread,
	// and the write to z follows the y read, y and z can share the same base address (in-place computing).
	if (beta == 0.0)
		z[outRow] = PREC_DMUL(alpha, zProd);
	else
		z[outRow] = PREC_DADD(PREC_DMUL (beta, yVal), PREC_DMUL (alpha, zProd));
}


__global__ void
spgpuDhellspmv_krn_ridx (double *z, const double *y, double alpha, const double* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, const int* rIdx, int rows, const double *x, double beta, int baseIndex)
{
	int i = threadIdx.x + blockIdx.x * (THREAD_BLOCK);
	if (i >= rows)
		return;

	int outRow = rIdx[i];
	double yVal;
	if (beta != 0.0)
		yVal = y[outRow];

	spgpuDhellspmv_ridx (i, yVal, outRow,
		z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rIdx, rows, x, beta, baseIndex);
}

__device__ void
spgpuDhellspmv_ (double *z, const double *y, double alpha, const double* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, int rows, const double *x, double beta, int baseIndex)
{
	int i = threadIdx.x + blockIdx.x * (THREAD_BLOCK);
	if (i >= rows)
		return;

	double yVal;

	if (beta != 0.0)
		yVal = y[i];

	spgpuDhellspmv_ridx (i, yVal, i,
		z, y, alpha, cM, rP, hackSize, hackOffsets, rS, NULL, rows, x, beta, baseIndex);
}

// Force to recompile and optimize with llvm
__global__ void
spgpuDhellspmv_krn_b0 (double *z, const double *y, double alpha, const double* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, int rows, const double *x, int baseIndex)
{
	spgpuDhellspmv_ (z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rows, x, 0.0f, baseIndex);
}

__global__ void
spgpuDhellspmv_krn (double *z, const double *y, double alpha, const double* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, int rows, const double *x, double beta, int baseIndex)
{
	spgpuDhellspmv_ (z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rows, x, beta, baseIndex);
}

void
_spgpuDhellspmv (spgpuHandle_t handle, double* z, const double *y, double alpha, const double* cM, const int* rP, int hackSize, const int* hackOffsets, const int* rS, const int* rIdx, int rows, const double *x, double beta, int baseIndex)
{
	dim3 block (THREAD_BLOCK);
	dim3 grid ((rows + THREAD_BLOCK - 1) / THREAD_BLOCK);

        int shrMemSize;
#if __CUDA_ARCH__ < 300
       	int warpsPerBlock = THREAD_BLOCK/handle->warpSize;
        shrMemSize = warpsPerBlock*sizeof(int);
#else
       	shrMemSize = 0;
#endif

#ifdef ENABLE_CACHE
	bind_tex_x ((const int2 *) x);
#endif

	if (rIdx)
		spgpuDhellspmv_krn_ridx <<< grid, block, shrMemSize, handle->currentStream >>> (z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rIdx, rows, x, beta, baseIndex);
	else
	{
		if (beta != 0.0)
			spgpuDhellspmv_krn <<< grid, block, shrMemSize, handle->currentStream >>> (z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rows, x, beta, baseIndex);
		else
			spgpuDhellspmv_krn_b0 <<< grid, block, shrMemSize, handle->currentStream >>> (z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rows, x, baseIndex);
	}

	
#ifdef ENABLE_CACHE
  	unbind_tex_x ((const int2 *) x);
#endif

}



void 
spgpuDhellspmv (spgpuHandle_t handle,
	__device double *z, 
	const __device double *y, 
	double alpha, 
	const __device double* cM, 
	const __device int* rP,
	int hackSize,
	const __device int* hackOffsets, 
	const __device int* rS,
	const __device int* rIdx, 
	int rows, 
	const __device double *x, 
	double beta,
	int baseIndex)
{
	__assert(hackSize % 32 == 0, "Only hacks whose length is a multiple of 32 are supported...");

	
	while (rows > MAX_N_FOR_A_CALL) //managing large vectors
	{
		_spgpuDhellspmv (handle, z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rIdx, MAX_N_FOR_A_CALL, x, beta, baseIndex);

		y = y + MAX_N_FOR_A_CALL;
		z = z + MAX_N_FOR_A_CALL;
		rS = rS + MAX_N_FOR_A_CALL;
		hackOffsets += MAX_N_FOR_A_CALL/hackSize;
		
		rows -= MAX_N_FOR_A_CALL;
	}
	
	_spgpuDhellspmv (handle, z, y, alpha, cM, rP, hackSize, hackOffsets, rS, rIdx, rows, x, beta, baseIndex);
	
	cudaCheckError("CUDA error on hell_dspmv");
}
