#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "stdio.h"
#include "cudalang.h"
#include "cudadebug.h"

extern "C"
{
#include "core.h"
#include "vector.h"
}



void spgpuDmamax(hipblasHandle_t handle, double *y, int n, __device double *x, int count, int pitch)
{
  int i,j;
  for (i=0; i < count; ++i)
    {
      //fprintf(stderr," Calling hipblasIdamax %d %p, %d\n",n,x,count);
      hipblasIdamax(handle,n,x,1,&j);
      //fprintf(stderr," Exit from  hipblasIdamax %d %p\n",j,x);
      hipError_t err = hipMemcpy(&(y[i]), &(x[j-1]), sizeof(double), hipMemcpyDeviceToHost);
      y[i] = fabs(y[i]);
      x += pitch;
    }
}
