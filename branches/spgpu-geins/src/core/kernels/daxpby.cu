#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"

extern "C"
{
#include "core.h"
#include "vector.h"
}


#include "debug.h"

#define BLOCK_SIZE 512

__global__ void spgpuDaxpby_krn(double *z, int n, double beta, double *y, double alpha, double* x)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	
	if (id < n)
	{
		// Since z, x and y are accessed with the same offset by the same thread,
		// and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).

		if (beta == 0.0)
			z[id] = PREC_DMUL(alpha,x[id]);
		else
			z[id] = PREC_DADD(PREC_DMUL(alpha, x[id]), PREC_DMUL(beta,y[id]));
	}
}


void spgpuDaxpby_(spgpuHandle_t handle,
	__device double *z,
	int n,
	double beta,
	__device double *y,
	double alpha,
	__device double* x)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;

	dim3 block(BLOCK_SIZE);
	dim3 grid(msize);

	spgpuDaxpby_krn<<<grid, block, 0, handle->currentStream>>>(z, n, beta, y, alpha, x);
}

void spgpuDaxpby(spgpuHandle_t handle,
	__device double *z,
	int n,
	double beta,
	__device double *y,
	double alpha,
	__device double* x)
{
	int maxNForACall = max(handle->maxGridSizeX, BLOCK_SIZE*handle->maxGridSizeX);
	while (n > maxNForACall) //managing large vectors
	{
		spgpuDaxpby_(handle, z, maxNForACall, beta, y, alpha, x);
		
		x = x + maxNForACall;
		y = y + maxNForACall;
		z = z + maxNForACall;
		n -= maxNForACall;
	}
	
	spgpuDaxpby_(handle, z, n, beta, y, alpha, x);

	cudaCheckError("CUDA error on daxpby");
}

void spgpuDmaxpby(spgpuHandle_t handle,
		  __device double *z,
		  int n,
		  double beta,
		  __device double *y,
		  double alpha,
		  __device double* x, 
		  int count, int pitch)
{

  for (int i=0; i<count; i++)
    spgpuDaxpby(handle, z+pitch*i, n, beta, y+pitch*i, alpha, x+pitch*i);
  
}
