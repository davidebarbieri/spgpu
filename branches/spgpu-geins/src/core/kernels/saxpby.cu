#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */
#include "cudadebug.h"
#include "cudalang.h"

extern "C"
{
#include "core.h"
#include "vector.h"
}


#include "debug.h"

#define BLOCK_SIZE 512

__global__ void spgpuSaxpby_krn(float *z, int n, float beta, float *y, float alpha, float* x)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	
	if (id < n)
	{
		// Since z, x and y are accessed with the same offset by the same thread,
		// and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).

		if (beta == 0.0f)
			z[id] = PREC_FMUL(alpha,x[id]);
		else
			z[id] = PREC_FADD(PREC_FMUL(alpha, x[id]), PREC_FMUL(beta,y[id]));
	}
}


void spgpuSaxpby_(spgpuHandle_t handle,
	__device float *z,
	int n,
	float beta,
	__device float *y,
	float alpha,
	__device float* x)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;

	dim3 block(BLOCK_SIZE);
	dim3 grid(msize);

	spgpuSaxpby_krn<<<grid, block, 0, handle->currentStream>>>(z, n, beta, y, alpha, x);
}

void spgpuSaxpby(spgpuHandle_t handle,
	__device float *z,
	int n,
	float beta,
	__device float *y,
	float alpha,
	__device float* x)
{
	int maxNForACall = max(handle->maxGridSizeX, BLOCK_SIZE*handle->maxGridSizeX);
	while (n > maxNForACall) //managing large vectors
	{
		spgpuSaxpby_(handle, z, maxNForACall, beta, y, alpha, x);
		
		x = x + maxNForACall;
		y = y + maxNForACall;
		z = z + maxNForACall;
		n -= maxNForACall;
	}
	
	spgpuSaxpby_(handle, z, n, beta, y, alpha, x);

	cudaCheckError("CUDA error on saxpby");
}

void spgpuSmaxpby(spgpuHandle_t handle,
		  __device float *z,
		  int n,
		  float beta,
		  __device float *y,
		  float alpha,
		  __device float* x, 
		  int count, int pitch)
{

  for (int i=0; i<count; i++)
    spgpuSaxpby(handle, z+pitch*i, n, beta, y+pitch*i, alpha, x+pitch*i);
  
}
